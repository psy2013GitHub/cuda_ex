#include "hip/hip_runtime.h"

//Udacity HW 4
//Radix Sorting

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>
#include <math.h>
/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */
__global__
void gen_hist(unsigned int * const d_inputVals, 
              unsigned int * d_hist, 
              unsigned int pass, 
              unsigned int numElems) {
   unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx >= numElems)
       return;
   unsigned int b = d_inputVals[idx] & (1u << pass);
   if (b) 
      atomicAdd(&(d_hist[1]),1);
   else
      atomicAdd(&(d_hist[0]),1); 
}    

__global__
void scan_ele(unsigned int* d_inputVals, unsigned int* d_scaned, unsigned int base, unsigned int pass, unsigned int numElems, unsigned int threadSize) {
    unsigned int idx = base*threadSize + threadIdx.x;
    if (idx >= numElems)
        return;
    unsigned int b = (d_inputVals[idx] & (1u << pass))?1:0;
    int spot, val;
    
    d_scaned[idx] = b;
    __syncthreads();
    for (unsigned int s=threadSize>>1; s>0; s=s>>1) {
        spot = idx - s;
        if (spot >= 0 && spot>=base*threadSize)
            val = d_scaned[spot];
        __syncthreads();
        if (spot >= 0 && spot>=base*threadSize)
            d_scaned[idx] += val;
        __syncthreads();
    }
    if (base>0)
          d_scaned[idx] +=  d_scaned[base*threadSize-1];
}    

/* scan_ele0, 错误方法 */
__global__
void scan_ele0(unsigned int* d_inputVals, unsigned int* d_scaned, unsigned int pass, unsigned int numElems) {
    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx >= numElems)
        return;
    unsigned int b = (d_inputVals[idx] & (1u << pass))?1:0;
    int spot, val;
    
    d_scaned[idx] = b;
    __syncthreads();
    for (unsigned int s=blockDim.x>>1; s>0; s=s>>1) {
        spot = idx - s;
        if (spot >= 0)
            val = d_scaned[spot];
        __syncthreads();
        if (spot >= 0)
            d_scaned[idx] += val;
        __syncthreads();
    }
    /* if (base>0)
          d_scaned[idx] +=  d_scaned[base*threadSize-1]; */
}    


__global__
void move_ele(unsigned int* const d_inputVals, 
              unsigned int* const d_inputPos,
              unsigned int* const d_outputVals,
              unsigned int* const d_outputPos,
              unsigned int* const d_scaned,
              unsigned int* const d_hist,
              unsigned int pass,
              unsigned int numElems) {
   unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx >= numElems)
       return;
    
   unsigned int b = d_inputVals[idx] & (1u << pass); 
   
   unsigned int base=0;
   unsigned int offset=0;
   unsigned int p=0; 
   if (b) {
      base = d_hist[0]; 
      offset = idx?d_scaned[idx-1]:0; // !!!
      // offset = d_scaned[idx]; // !!! 
   } else {
      base = 0;
      offset = idx - d_scaned[idx];
   }
   p = base + offset; 
   d_outputVals[p] = d_inputVals[idx];
   d_outputPos[p]  = d_inputPos[idx];  
}    

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
    
 dim3 gridSize(ceil((float)(numElems)/1024.0f)+1);
 dim3 blockSize(1024);
 
 unsigned int *d_hist;
 hipMalloc((void **)(&d_hist), sizeof(unsigned int) * 2);
 
 size_t bytes = sizeof(unsigned int) * numElems;   
    
 /* unsigned int *d_inputVals_tmp;
 hipMalloc((void **)(&d_inputVals_tmp), bytes);
 hipMemcpy(d_inputVals_tmp, d_inputVals, bytes, hipMemcpyDeviceToDevice);   
 unsigned int *d_inputPos_tmp;
 hipMalloc((void **)(&d_inputPos_tmp), bytes);
 hipMemcpy(d_inputPos_tmp,  d_inputPos, bytes, hipMemcpyDeviceToDevice);     
 */
    
 /* unsigned int *d_scan;
 checkCudaErrors(hipMalloc((void **)(&d_scan), bytes)); */
 unsigned int *d_scaned;
 checkCudaErrors(hipMalloc((void **)(&d_scaned), bytes));
 // unsigned int h_scaned[numElems]; 
    
 for (int pass=0; pass<32; pass++) {
     
     checkCudaErrors(hipMemset(d_hist, 0, sizeof(unsigned int) << 1));
     checkCudaErrors(hipMemset(d_scaned, 0, bytes));
     checkCudaErrors(hipMemset(d_outputVals, 0, bytes));
     checkCudaErrors(hipMemset(d_outputPos, 0, bytes));
     
     gen_hist<<<gridSize, blockSize>>>(d_inputVals, d_hist, pass, numElems);
     hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
     
     // 为了对整个数组进行排序，只能一个block一个block处理，因为scan调用了__syncthreads()，只能保证在block内有效
     for (unsigned base=0; base < gridSize.x; ++base) { // loop each block
         scan_ele<<<dim3(1), blockSize>>>(d_inputVals, d_scaned, base, pass, numElems, blockSize.x);
         hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
     }
     /* 以下方法错误，无法对整个进行数组排序
     scan_ele0<<<gridSize, blockSize>>>(d_inputVals, d_scaned, pass, numElems);
     hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
     */
     move_ele<<<gridSize, blockSize>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos, d_scaned, d_hist, pass, numElems);
     hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
     
     checkCudaErrors(hipMemcpy(d_inputVals, d_outputVals, bytes, hipMemcpyDeviceToDevice));    
     checkCudaErrors(hipMemcpy(d_inputPos,  d_outputPos,  bytes, hipMemcpyDeviceToDevice));
     hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
 }    
 /*
 checkCudaErrors(hipMemcpy(d_inputVals, d_inputVals_tmp, bytes, hipMemcpyDeviceToDevice));
 checkCudaErrors(hipMemcpy(d_inputPos,  d_inputPos_tmp,  bytes, hipMemcpyDeviceToDevice));   
 */   
 checkCudaErrors(hipFree((void *)(d_hist)));
 /*
 checkCudaErrors(hipFree((void *)(d_inputVals_tmp)));
 checkCudaErrors(hipFree((void *)(d_inputPos_tmp)));   
 */   
}

